#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include "icp_kernel.h"
#include "svd3.h"

extern int numPoints;
extern int numDataPoints;
extern int numModelPoints;

extern glm::vec3* dev_pos;
extern glm::vec3* dev_col;

extern glm::vec3* dev_dataBuffer;
extern glm::vec3* dev_modelBuffer;
extern glm::vec3* dev_corrBuffer;

extern glm::vec3* dev_centeredCorrBuffer;
extern glm::vec3* dev_centeredDataBuffer;
extern glm::mat3* dev_ABtBuffer;

//Helper functions
void matSVD(glm::mat3& ABt, glm::mat3& U, glm::mat3& S, glm::mat3& V)
{
	svd(ABt[0].x, ABt[0].y, ABt[0].z,
		ABt[1].x, ABt[1].y, ABt[1].z,
		ABt[2].x, ABt[2].y, ABt[2].z,

		U[0].x, U[0].y, U[0].z,
		U[1].x, U[1].y, U[1].z,
		U[2].x, U[2].y, U[2].z,

		S[0].x, S[0].y, S[0].z,
		S[1].x, S[1].y, S[1].z,
		S[2].x, S[2].y, S[2].z,

		V[0].x, V[0].y, V[0].z,
		V[1].x, V[1].y, V[1].z,
		V[2].x, V[2].y, V[2].z);
}

// CPU ICP pipeline
void ICP::CPUStep(std::vector<glm::vec3>& dataBuffer, std::vector<glm::vec3>& modelBuffer) {

	// Find nearest correspondences
	std::vector<glm::vec3> corrBuffer(numDataPoints);
	for (int i = 0; i < numDataPoints; i++) {
		float distMin = FLT_MAX;
		for (int j = 0; j < numModelPoints; j++) {
			float dist = glm::distance(dataBuffer[i], modelBuffer[j]);
			if (distMin > dist) {
				distMin = dist;
				corrBuffer[i] = modelBuffer[j];
			}
		}
	}

	// Centralize
	glm::vec3 meanData(0.0f);
	glm::vec3 meanCorr(0.0f);
	for (int i = 0; i < numDataPoints; i++) {
		meanData += dataBuffer[i];
		meanCorr += corrBuffer[i];
	}
	meanData /= static_cast<float>(numDataPoints);
	meanCorr /= static_cast<float>(numDataPoints);

	std::vector<glm::vec3> centeredDataBuffer(numDataPoints);
	std::vector<glm::vec3> centeredCorrBuffer(numDataPoints);
	for (int i = 0; i < numDataPoints; i++) {
		centeredDataBuffer[i] = dataBuffer[i] - meanData;
		centeredCorrBuffer[i] = corrBuffer[i] - meanCorr;
	}

	// Calculating rotation and translations
	// PnP algorithm: minimizing A-RB equals to minimizing R-AB^T
	// Kabsch algorthm: Orthogonalize the rotation matrix with SVD: AB^T = USV^T, R = UV^T
	glm::mat3 ABt(0.0f);
	for (int i = 0; i < numDataPoints; i++) {
		ABt += glm::outerProduct(centeredDataBuffer[i], centeredCorrBuffer[i]);
	}

	//compute SVD of ABt
	glm::mat3 R(0.0f), U(0.0f), S(0.0f), V(0.0f);
	glm::vec3 T(0.0f);

	matSVD(ABt, U, S, V);

	R = glm::transpose(U) * V; // Strange glm::mat column sequence >:(
	T = meanCorr - (R * meanData);

	// Update and draw
	for (int i = 0; i < numDataPoints; i++)
		dataBuffer[i] = R * dataBuffer[i] + T;
	std::copy(dataBuffer.begin(), dataBuffer.end(), dev_pos + numModelPoints);
	hipDeviceSynchronize();
}

// Helper kernel functions
__global__ void kernSearchNearest(int numDataPoints, int numModelPoints,
	const glm::vec3* dataBuffer, const glm::vec3* modelBuffer, glm::vec3* corrBuffer) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < numDataPoints) {
		float distMin = FLT_MAX;
		for (int j = 0; j < numModelPoints; j++) {
			float dist = glm::distance(dataBuffer[index], modelBuffer[j]);
			if (distMin > dist) {
				distMin = dist;
				corrBuffer[index] = modelBuffer[j];
			}
		}
	}
}

__global__ void kernCentralize(int numDataPoints, glm::vec3* in, glm::vec3* out, glm::vec3 mean) {

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= numDataPoints) return;

	out[index] = in[index] - mean;
}

__global__ void kernOuterProduct(int numDataPoints,
	glm::vec3* A, glm::vec3* B, glm::mat3* out) {

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= numDataPoints) return;

	out[index] = glm::outerProduct(A[index], B[index]);
}

__global__ void kernTransform(int numDataPoints, glm::vec3* pos, glm::mat3 R, glm::vec3 T) {

	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index >= numDataPoints) return;

	pos[index] = R * pos[index] + T;
}


// GPU ICP Pipeline
void ICP::naiveGPUStep() {

	dim3 dataBlocksPerGrid((numDataPoints + blockSize - 1) / blockSize);
	// Find nearest correspondences
	kernSearchNearest << <dataBlocksPerGrid, blockSize >> > (numDataPoints, numModelPoints, dev_dataBuffer, dev_modelBuffer, dev_corrBuffer);
	hipDeviceSynchronize();

	// Centralize
	glm::vec3 meanData = thrust::reduce(dev_dataBuffer, dev_dataBuffer + numDataPoints);
	glm::vec3 meanCorr = thrust::reduce(dev_corrBuffer, dev_corrBuffer + numDataPoints);
	meanData = meanData / static_cast<float>(numDataPoints);
	meanCorr = meanCorr / static_cast<float>(numDataPoints);

	kernCentralize << < dataBlocksPerGrid, blockSize >> > (numDataPoints, dev_dataBuffer, dev_centeredDataBuffer, meanData);
	kernCentralize << < dataBlocksPerGrid, blockSize >> > (numDataPoints, dev_corrBuffer, dev_centeredCorrBuffer, meanCorr);
	hipDeviceSynchronize();

	// Calculating rotation and translations
	// PnP algorithm: minimizing A-RB equals to minimizing R-AB^T
	// Kabsch algorthm: Orthogonalize the rotation matrix with SVD: AB^T = USV^T, R = UV^T
	kernOuterProduct << <dataBlocksPerGrid, blockSize >> > (numDataPoints, dev_centeredDataBuffer, dev_centeredCorrBuffer, dev_ABtBuffer);
	hipDeviceSynchronize();

	glm::mat3 ABt = thrust::reduce(dev_ABtBuffer, dev_ABtBuffer + numDataPoints);

	//compute SVD of ABt
	glm::mat3 R(0.0f), U(0.0f), S(0.0f), V(0.0f);
	glm::vec3 T(0.0f);

	matSVD(ABt, U, S, V);

	R = glm::transpose(U) * V; // Strange glm::mat column sequence >:(
	T = meanCorr - (R * meanData);

	// Update and draw
	kernTransform << < dataBlocksPerGrid, blockSize >> > (numDataPoints, dev_dataBuffer, R, T);
	hipDeviceSynchronize();

	std::copy(&dev_dataBuffer[0], &dev_dataBuffer[0] + numDataPoints, &dev_pos[numModelPoints]);
	hipDeviceSynchronize();
}
