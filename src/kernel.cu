#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include "kernel.h"

extern int mode;
extern int numPoints;
extern int numDataPoints;
extern int numModelPoints;

extern glm::vec3* dev_pos;
extern glm::vec3* dev_col;
extern glm::vec3* dev_dataBuffer;
extern glm::vec3* dev_modelBuffer;
extern glm::vec3* dev_corrBuffer;
extern glm::vec3* dev_centeredCorrBuffer;
extern glm::vec3* dev_centeredDataBuffer;
extern glm::mat3* dev_ABtBuffer;

extern glm::vec3* dev_optDataBuffer;
extern glm::vec3* dev_curDataBuffer;

extern FlattenedKDTree* dev_fkdt;
extern float* dev_minDists;
extern size_t* dev_minIndices;

extern int numCubes;
extern std::vector<glm::vec3> transCubePosBuffer;
extern std::vector<glm::vec3> transCubeColBuffer;
extern std::vector<glm::vec3> rotCubePosBuffer;
extern std::vector<glm::vec3> rotCubeColBuffer;
extern glm::vec3* dev_cubePosBuffer;
extern glm::vec3* dev_cubeColBuffer;
extern glm::vec3* dev_rotCubePosBuffer;
extern glm::vec3* dev_rotCubeColBuffer;

extern int maxTNodes;
extern float* dev_errors;
extern float* dev_rot_ub_trans_ub;
extern float* dev_rot_ub_trans_lb;


// Helper Functions
void checkCUDAError(const char* msg, int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// Helper kernel functions
__global__ void kernResetVec3Buffer(int N, glm::vec3* intBuffer, glm::vec3 value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		intBuffer[index] = value;
	}
}

__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyColorsToVBO(int N, glm::vec3* col, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = col[index].x;
		vbo[4 * index + 1] = col[index].y;
		vbo[4 * index + 2] = col[index].z;
		vbo[4 * index + 3] = 1.0f;
	}
}

void PointCloud::initBuffers(std::vector<glm::vec3>& dataBuffer, std::vector<glm::vec3>& modelBuffer) {
	// Use unified memory
	hipDeviceSynchronize();
	hipMallocManaged((void**)&dev_pos, numPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_pos failed!");
	hipMallocManaged((void**)&dev_col, numPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_col failed!");
	hipMallocManaged((void**)&dev_dataBuffer, numDataPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_dataBuffer failed!");
	hipMallocManaged((void**)&dev_modelBuffer, numModelPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_modelBuffer failed!");
	hipMallocManaged((void**)&dev_corrBuffer, numDataPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_corrBuffer failed!");
	hipMallocManaged((void**)&dev_centeredDataBuffer, numDataPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_centeredDataBuffer failed!");
	hipMallocManaged((void**)&dev_centeredCorrBuffer, numDataPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_centeredCorrBuffer failed!");
	hipMallocManaged((void**)&dev_ABtBuffer, numDataPoints * sizeof(glm::mat3));
	checkCUDAErrorWithLine("hipMallocManaged dev_ABtBuffer failed!");
	hipMallocManaged((void**)&dev_minDists, numDataPoints * sizeof(float));
	checkCUDAErrorWithLine("hipMallocManaged dev_dataBuffer failed!");
	hipMallocManaged((void**)&dev_minIndices, numModelPoints * sizeof(size_t));
	checkCUDAErrorWithLine("hipMallocManaged dev_modelBuffer failed!");

	if (mode == GOICP_CPU)
	{
		hipMallocManaged((void**)&dev_optDataBuffer, numDataPoints * sizeof(glm::vec3));
		checkCUDAErrorWithLine("hipMallocManaged dev_optDataBuffer failed!");
		hipMallocManaged((void**)&dev_curDataBuffer, numDataPoints * sizeof(glm::vec3));
		checkCUDAErrorWithLine("hipMallocManaged dev_curDataBuffer failed!");
	}

	if (mode == GOICP_GPU)
	{
		hipMallocManaged((void**)&dev_errors, sizeof(float) * numDataPoints);
		checkCUDAErrorWithLine("hipMallocManaged dev_errors failed!");
		hipMallocManaged((void**)&dev_rot_ub_trans_ub, sizeof(float) * numDataPoints * maxTNodes);
		checkCUDAErrorWithLine("hipMallocManaged dev_rot_ub_trans_ub failed!");
		hipMallocManaged((void**)&dev_rot_ub_trans_lb, sizeof(float) * numDataPoints * maxTNodes);
		checkCUDAErrorWithLine("hipMallocManaged dev_rot_ub_trans_lb failed!");

		hipMallocManaged((void**)&dev_cubePosBuffer, 2 * numCubes * sizeof(glm::vec3));
		checkCUDAErrorWithLine("hipMallocManaged dev_cubePosBuffer failed!");
		hipMallocManaged((void**)&dev_cubeColBuffer, 2 * numCubes * sizeof(glm::vec3));
		checkCUDAErrorWithLine("hipMallocManaged dev_transCubeSizeBuffer failed!");
	}

	// Set Posistion Buffer
	std::copy(dataBuffer.begin(), dataBuffer.end(), dev_dataBuffer);
	std::copy(modelBuffer.begin(), modelBuffer.end(), dev_modelBuffer);
	std::copy(modelBuffer.begin(), modelBuffer.end(), dev_pos);
	std::copy(dataBuffer.begin(), dataBuffer.end(), dev_pos + numModelPoints);
	if (mode == GOICP_CPU) 
		std::copy(dataBuffer.begin(), dataBuffer.end(), dev_pos + numModelPoints + numDataPoints);

	// Set color buffer
	dim3 dataBlocksPerGrid((numDataPoints + blockSize - 1) / blockSize);
	dim3 modelBlocksPerGrid((numModelPoints + blockSize - 1) / blockSize);
	kernResetVec3Buffer << <modelBlocksPerGrid, blockSize >> > (numModelPoints, dev_col, glm::vec3(0, 0, 1));
	kernResetVec3Buffer << < dataBlocksPerGrid, blockSize >> > (numDataPoints, &dev_col[numModelPoints], glm::vec3(1, 0, 0));
	kernResetVec3Buffer << <modelBlocksPerGrid, blockSize >> > (numModelPoints, dev_col, glm::vec3(0, 0, 1));
	if (mode == GOICP_CPU) 
		kernResetVec3Buffer <<< dataBlocksPerGrid, blockSize >> > (numDataPoints, &dev_col[numModelPoints + numDataPoints], glm::vec3(1.0));
	hipDeviceSynchronize();

	// Set search buffer
	if (mode == GOICP_GPU)
	{
		std::copy(transCubePosBuffer.begin(), transCubePosBuffer.end(), dev_cubePosBuffer);
		std::copy(transCubeColBuffer.begin(), transCubeColBuffer.end(), dev_cubeColBuffer);
		std::copy(rotCubePosBuffer.begin(), rotCubePosBuffer.end(), dev_cubePosBuffer + numCubes);
		std::copy(rotCubeColBuffer.begin(), rotCubeColBuffer.end(), dev_cubeColBuffer + numCubes);
		hipDeviceSynchronize();
	}
}

void PointCloud::copyPointsToVBO(int N, glm::vec3* posBuffer, glm::vec3* colBuffer, float* vbodptr_positions, float* vbodptr_colors) {
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);
	kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> > (N, posBuffer, vbodptr_positions, scene_scale);
	kernCopyColorsToVBO << <fullBlocksPerGrid, blockSize >> > (N, colBuffer, vbodptr_colors, scene_scale);
	checkCUDAErrorWithLine("copyPointsToVBO failed!");
	hipDeviceSynchronize();
}

void PointCloud::cleanupBuffers() {
	hipFree(dev_pos);
	hipFree(dev_col);
	hipFree(dev_modelBuffer);
	hipFree(dev_dataBuffer);
	hipFree(dev_corrBuffer);
	hipFree(dev_centeredDataBuffer);
	hipFree(dev_centeredCorrBuffer);
	hipFree(dev_ABtBuffer);
	hipFree(dev_fkdt);
	hipFree(dev_minDists);
	hipFree(dev_minIndices);
	hipFree(dev_cubePosBuffer);
	hipFree(dev_cubeColBuffer);
	if (mode == GOICP_CPU)
	{
		hipFree(dev_optDataBuffer);
		hipFree(dev_curDataBuffer);
	}
	if (mode == GOICP_GPU)
	{
		hipFree(dev_errors);
		hipFree(dev_rot_ub_trans_ub);
		hipFree(dev_rot_ub_trans_lb);
		hipFree(dev_cubePosBuffer);
		hipFree(dev_cubeColBuffer);
	}

	checkCUDAErrorWithLine("hipFree failed!");
}

