#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include "kernel.h"


extern int numPoints;
extern int numDataPoints;
extern int numModelPoints;

extern glm::vec3* dev_pos;
extern glm::vec3* dev_col;

extern glm::vec3* dev_dataBuffer;
extern glm::vec3* dev_modelBuffer;
extern glm::vec3* dev_corrBuffer;

extern glm::vec3* dev_centeredCorrBuffer;
extern glm::vec3* dev_centeredDataBuffer;
extern glm::mat3* dev_ABtBuffer;

extern FlattenedKDTree* dev_fkdt;
extern float* dev_minDists;
extern size_t* dev_minIndices;

// Helper Functions
void checkCUDAError(const char* msg, int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

// Helper kernel functions
__global__ void kernResetVec3Buffer(int N, glm::vec3* intBuffer, glm::vec3 value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		intBuffer[index] = value;
	}
}

__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyColorsToVBO(int N, glm::vec3* col, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = col[index].x + 0.3f;
		vbo[4 * index + 1] = col[index].y + 0.3f;
		vbo[4 * index + 2] = col[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

void PointCloud::initBuffers(std::vector<glm::vec3>& dataBuffer, std::vector<glm::vec3>& modelBuffer) {
	// Use unified memory
	hipDeviceSynchronize();
	hipMallocManaged((void**)&dev_pos, numPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_pos failed!");
	hipMallocManaged((void**)&dev_col, numPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_col failed!");
	hipMallocManaged((void**)&dev_dataBuffer, numDataPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_dataBuffer failed!");
	hipMallocManaged((void**)&dev_modelBuffer, numModelPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_modelBuffer failed!");
	hipMallocManaged((void**)&dev_corrBuffer, numDataPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_corrBuffer failed!");
	hipMallocManaged((void**)&dev_centeredDataBuffer, numDataPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_centeredDataBuffer failed!");
	hipMallocManaged((void**)&dev_centeredCorrBuffer, numDataPoints * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMallocManaged dev_centeredCorrBuffer failed!");
	hipMallocManaged((void**)&dev_ABtBuffer, numDataPoints * sizeof(glm::mat3));
	checkCUDAErrorWithLine("hipMallocManaged dev_ABtBuffer failed!");
	hipMallocManaged((void**)&dev_minDists, numDataPoints * sizeof(float));
	checkCUDAErrorWithLine("hipMallocManaged dev_dataBuffer failed!");
	hipMallocManaged((void**)&dev_minIndices, numModelPoints * sizeof(size_t));
	checkCUDAErrorWithLine("hipMallocManaged dev_modelBuffer failed!");

	// Set Posistion Buffer
	std::copy(dataBuffer.begin(), dataBuffer.end(), dev_dataBuffer);
	std::copy(modelBuffer.begin(), modelBuffer.end(), dev_modelBuffer);
	std::copy(modelBuffer.begin(), modelBuffer.end(), dev_pos);
	std::copy(dataBuffer.begin(), dataBuffer.end(), dev_pos + numModelPoints);

	// Set color buffer
	dim3 dataBlocksPerGrid((numDataPoints + blockSize - 1) / blockSize);
	dim3 modelBlocksPerGrid((numModelPoints + blockSize - 1) / blockSize);
	kernResetVec3Buffer << <modelBlocksPerGrid, blockSize >> > (numModelPoints, dev_col, glm::vec3(0, 0, 1));
	kernResetVec3Buffer << < dataBlocksPerGrid, blockSize >> > (numDataPoints, &dev_col[numModelPoints], glm::vec3(1, 0, 0));
	kernResetVec3Buffer << <modelBlocksPerGrid, blockSize >> > (numModelPoints, dev_col, glm::vec3(0, 0, 1));
	hipDeviceSynchronize();
}

void PointCloud::copyPointsToVBO(float* vbodptr_positions, float* vbodptr_colors) {
	dim3 fullBlocksPerGrid((numPoints + blockSize - 1) / blockSize);
	kernCopyPositionsToVBO <<<fullBlocksPerGrid, blockSize >> > (numPoints, dev_pos, vbodptr_positions, scene_scale);
	kernCopyColorsToVBO << <fullBlocksPerGrid, blockSize >> > (numPoints, dev_col, vbodptr_colors, scene_scale);
	checkCUDAErrorWithLine("copyPointsToVBO failed!");

	hipDeviceSynchronize();
}

void PointCloud::cleanupBuffers() {
	hipFree(dev_pos);
	hipFree(dev_col);
	hipFree(dev_modelBuffer);
	hipFree(dev_dataBuffer);
	hipFree(dev_corrBuffer);
	hipFree(dev_centeredDataBuffer);
	hipFree(dev_centeredCorrBuffer);
	hipFree(dev_ABtBuffer);
	hipFree(dev_fkdt);
	hipFree(dev_minDists);
	hipFree(dev_minIndices);

	checkCUDAErrorWithLine("hipFree failed!");
}

